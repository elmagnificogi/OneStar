#include "CudaProcess.cuh"

// �ϐ�����
CudaInputMaster* cu_HostInputMaster;
_u32* cu_HostInputCoefficientData;
_u32* cu_HostInputSearchPattern;

// ���ʋ���
int* cu_HostResultCount;
_u64* cu_HostResult;

// �萔
const int c_SizeResult = 32;

// ������
void CudaInitializeImpl()
{
	// �z�X�g�������̊m��
	hipHostMalloc(&cu_HostInputMaster, sizeof(CudaInputMaster));
	hipHostMalloc(&cu_HostResultCount, sizeof(int));
	hipHostMalloc(&cu_HostResult, sizeof(_u64) * c_SizeResult);

	// �f�[�^�̏�����
	cu_HostInputMaster->ecBit = -1;
}

// �I��
void CudaFinalizeImpl()
{
	// �z�X�g���������
	hipHostFree(cu_HostResult);
	hipHostFree(cu_HostResultCount);
	hipHostFree(cu_HostInputMaster);
}
