#include "CudaProcess.cuh"
#include "Data.h"

// �萔
static CudaConst* cu_HostConstData;
CudaConst* cu_DeviceConstData;

// �ϐ�����
CudaInputMaster* cu_HostInputMaster;
_u32* cu_HostInputCoefficientData;
_u32* cu_HostInputSearchPattern;

// ���ʋ���
int* cu_HostResultCount;
_u64* cu_HostResult;

// �萔
const int c_SizeResult = 32;

// ������
void CudaInitializeImpl()
{
	// �z�X�g�������̊m��
	hipHostMalloc(&cu_HostConstData, sizeof(CudaConst));
	hipHostMalloc(&cu_HostInputMaster, sizeof(CudaInputMaster));
	hipHostMalloc(&cu_HostResultCount, sizeof(int));
	hipHostMalloc(&cu_HostResult, sizeof(_u64) * c_SizeResult);

	// �f�o�C�X�������̊m��
	hipMalloc(&cu_DeviceConstData, sizeof(CudaConst));

	// �f�[�^�̏�����
	cu_HostInputMaster->ecBit = -1;

	// �萔�f�[�^��]��
	cu_HostConstData->natureTable[0] = c_NatureTable[0];
	cu_HostConstData->natureTable[1] = c_NatureTable[1];
	cu_HostConstData->natureTable[2] = c_NatureTable[2];
	hipMemcpy(cu_DeviceConstData, cu_HostConstData, sizeof(CudaConst), hipMemcpyHostToDevice);
}

// �I��
void CudaFinalizeImpl()
{
	// �f�o�C�X���������
	hipFree(cu_DeviceConstData);

	// �z�X�g���������
	hipHostFree(cu_HostResult);
	hipHostFree(cu_HostResultCount);
	hipHostFree(cu_HostInputMaster);
	hipHostFree(cu_HostConstData);
}
