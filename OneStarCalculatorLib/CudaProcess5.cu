#include "hip/hip_runtime.h"
#include "CudaProcess5.cuh"
#include "Data.h"

//�f�o�C�X�������̃|�C���^
static CudaInputMaster* pDeviceInput;
static _u32* pDeviceCoefficientData;
static _u32* pDeviceSearchPattern;
static int* pDeviceResultCount;
static _u64* pDeviceResult;

// ������s�萔
const int c_SizeBlockX = 1024;
const int c_SizeGridX = 1024 * 512;
const int c_SizeResult = 32;

// �v�Z����J�[�l��
__global__ static void kernel_calc(
	CudaInputMaster* pSrc,
	_u32* pCoefficient,
	_u32* pSearchPattern,
	int* pResultCount,
	_u64* pResult,
	_u32 param)
{
	int targetId = (blockIdx.x / 16) * 1024 + threadIdx.x; // �ő�19bit - 4bit + 10bit = 25bit
	int chunkId = blockIdx.x % 16;

	param |= targetId; // 25bit

	_u32 targetUpper = 0;
	_u32 targetLower = 0;

	// ����25bit = �̒l
	targetUpper |= (param & 0x1F00000ul); // iv0_0
	targetLower |= ((param & 0x3E0ul) << 10); // iv3_0
	targetUpper |= ((param & 0xF8000ul) >> 5); // iv1_0
	targetLower |= ((param & 0x1Ful) << 5); // iv4_0
	targetUpper |= ((param & 0x7C00ul) >> 10); // iv2_0

	// �B���ꂽ�l�𐄒�
	targetUpper |= ((32ul + pSrc->ivs[0] - ((param & 0x1F00000ul) >> 20)) & 0x1F) << 15;
	targetLower |= ((32ul + pSrc->ivs[3] - ((param & 0x3E0ul) >> 5)) & 0x1F) << 10;
	targetUpper |= ((32ul + pSrc->ivs[1] - ((param & 0xF8000ul) >> 15)) & 0x1F) << 5;
	targetLower |= ((32ul + pSrc->ivs[4] - (param & 0x1Ful)) & 0x1F);
	targetLower |= ((32ul + pSrc->ivs[2] - ((param & 0x7C00ul) >> 10)) & 0x1F) << 20;

	// target�x�N�g�����͊���

	targetUpper ^= pSrc->constantTermVector[0];
	targetLower ^= pSrc->constantTermVector[1];

	// ���������L���b�V��

	__shared__ _u32 answerFlag[128];
	__shared__ _u32 coefficientData[1024 * 2];
	__shared__ _u32 searchPattern[1024];
	__shared__ PokemonData pokemon[4];
	__shared__ int ecBit;
	__shared__ bool ecMod[3][6];

	if(threadIdx.x % 8 == 0)
	{
		answerFlag[threadIdx.x / 8] = pSrc->answerFlag[threadIdx.x / 8];
	}
	else if(threadIdx.x % 8 == 1)
	{
		pokemon[0] = pSrc->pokemon[0];
	}
	else if(threadIdx.x % 8 == 2)
	{
		pokemon[1] = pSrc->pokemon[1];
	}
	else if(threadIdx.x % 8 == 3)
	{
		pokemon[2] = pSrc->pokemon[2];
	}
	else if(threadIdx.x % 8 == 4)
	{
		pokemon[3] = pSrc->pokemon[3];
	}
	else if(threadIdx.x % 8 == 5)
	{
		ecBit = pSrc->ecBit;
	}
	else if(threadIdx.x % 8 == 6)
	{
		ecMod[0][0] = pSrc->ecMod[0][0];
		ecMod[0][1] = pSrc->ecMod[0][1];
		ecMod[0][2] = pSrc->ecMod[0][2];
		ecMod[0][3] = pSrc->ecMod[0][3];
		ecMod[0][4] = pSrc->ecMod[0][4];
		ecMod[0][5] = pSrc->ecMod[0][5];
		ecMod[1][0] = pSrc->ecMod[1][0];
		ecMod[1][1] = pSrc->ecMod[1][1];
		ecMod[1][2] = pSrc->ecMod[1][2];
	}
	else if(threadIdx.x % 8 == 7)
	{
		ecMod[1][3] = pSrc->ecMod[1][3];
		ecMod[1][4] = pSrc->ecMod[1][4];
		ecMod[1][5] = pSrc->ecMod[1][5];
		ecMod[2][0] = pSrc->ecMod[2][0];
		ecMod[2][1] = pSrc->ecMod[2][1];
		ecMod[2][2] = pSrc->ecMod[2][2];
		ecMod[2][3] = pSrc->ecMod[2][3];
		ecMod[2][4] = pSrc->ecMod[2][4];
		ecMod[2][5] = pSrc->ecMod[2][5];
	}
	coefficientData[threadIdx.x * 2] = pCoefficient[chunkId * 2048 + threadIdx.x * 2];
	coefficientData[threadIdx.x * 2 + 1] = pCoefficient[chunkId * 2048 + threadIdx.x * 2 + 1];
	searchPattern[threadIdx.x] = pSearchPattern[chunkId * 1024 + threadIdx.x];

	__syncthreads();

	_u32 processedTargetUpper = 0;
	_u32 processedTargetLower = 0;
	for(int i = 0; i < 32; ++i)
	{
		processedTargetUpper |= (CudaGetSignature(answerFlag[i * 2] & targetUpper) ^ CudaGetSignature(answerFlag[i * 2 + 1] & targetLower)) << (31 - i);
		processedTargetLower |= (CudaGetSignature(answerFlag[(i + 32) * 2] & targetUpper) ^ CudaGetSignature(answerFlag[(i + 32) * 2 + 1] & targetLower)) << (31 - i);
	}

	_u32 seeds[7]; // S0Upper�AS0Lower�AS1Upper�AS1Lower
	_u32 next[7]; // S0Upper�AS0Lower�AS1Upper�AS1Lower
	_u64 temp64;
	_u32 temp32;
	for(int i = 0; i < 1024; ++i)
	{
		seeds[0] = processedTargetUpper ^ coefficientData[i * 2];
		seeds[1] = processedTargetLower ^ coefficientData[i * 2 + 1] | searchPattern[i];

		// ��`�ӏ�

		if(ecBit >= 0 && (seeds[1] & 1) != ecBit)
		{
			continue;
		}

		temp64 = ((_u64)seeds[0] << 32 | seeds[1]) + 0x82a2b175229d6a5bull;

		seeds[2] = 0x82a2b175ul;
		seeds[3] = 0x229d6a5bul;

		next[0] = (_u32)(temp64 >> 32);
		next[1] = (_u32)temp64;
		next[2] = 0x82a2b175ul;
		next[3] = 0x229d6a5bul;

		temp64 = ((_u64)seeds[0] << 32 | seeds[1]);

		// ��������i�荞��

		// EC
		temp32 = CudaNext(seeds, 0xFFFFFFFFu);
		// 1�C�ڌ�
		if(ecMod[0][temp32 % 6] == false)
		{
			continue;
		}
		// 2�C�ڌ�
		if(ecMod[1][temp32 % 6] == false)
		{
			continue;
		}

		// EC
		temp32 = CudaNext(next, 0xFFFFFFFFu);
		// 3�C�ڌ�
		if(ecMod[2][temp32 % 6] == false)
		{
			continue;
		}

		// 2�C�ڂ��Ƀ`�F�b�N
		CudaNext(next); // OTID
		CudaNext(next); // PID

		{
			int ivs[6] = { -1, -1, -1, -1, -1, -1 };
			temp32 = 0;
			do {
				int fixedIndex = 0;
				do {
					fixedIndex = CudaNext(next, 7); // V�ӏ�
				} while(fixedIndex >= 6);

				if(ivs[fixedIndex] == -1)
				{
					ivs[fixedIndex] = 31;
					++temp32;
				}
			} while(temp32 < pokemon[2].flawlessIvs);

			// �̒l
			temp32 = 1;
			for(int i = 0; i < 6; ++i)
			{
				if(ivs[i] == 31)
				{
					if(pokemon[2].ivs[i] != 31)
					{
						temp32 = 0;
						break;
					}
				}
				else if(pokemon[2].ivs[i] != CudaNext(next, 0x1F))
				{
					temp32 = 0;
					break;
				}
			}
			if(temp32 == 0)
			{
				continue;
			}

			// ����
			temp32 = 0;
			if(pokemon[2].abilityFlag == 3)
			{
				temp32 = CudaNext(next, 1);
			}
			else
			{
				do {
					temp32 = CudaNext(next, 3);
				} while(temp32 >= 3);
			}
			if((pokemon[2].ability >= 0 && pokemon[2].ability != temp32) || (pokemon[2].ability == -1 && temp32 >= 2))
			{
				continue;
			}

			// ���ʒl
			if(!pokemon[2].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = CudaNext(next, 0xFF);
				} while(temp32 >= 253);
			}

			// ���i
			temp32 = 0;
			do {
				temp32 = CudaNext(next, 0x1F);
			} while(temp32 >= 25);

			if(temp32 != pokemon[2].nature)
			{
				continue;
			}
		}

		// 1�C��
		CudaNext(seeds); // OTID
		CudaNext(seeds); // PIT

		{
			// ��Ԃ�ۑ�
			next[0] = seeds[0];
			next[1] = seeds[1];
			next[2] = seeds[2];
			next[3] = seeds[3];

			{
				int ivs[6] = { -1, -1, -1, -1, -1, -1 };
				temp32 = 0;
				do {
					int fixedIndex = 0;
					do {
						fixedIndex = CudaNext(seeds, 7); // V�ӏ�
					} while(fixedIndex >= 6);

					if(ivs[fixedIndex] == -1)
					{
						ivs[fixedIndex] = 31;
						++temp32;
					}
				} while(temp32 < pokemon[0].flawlessIvs);

				// �̒l
				temp32 = 1;
				for(int i = 0; i < 6; ++i)
				{
					if(ivs[i] == 31)
					{
						if(pokemon[0].ivs[i] != 31)
						{
							temp32 = 0;
							break;
						}
					}
					else if(pokemon[0].ivs[i] != CudaNext(seeds, 0x1F))
					{
						temp32 = 0;
						break;
					}
				}
				if(temp32 == 0)
				{
					continue;
				}
			}
			{
				int ivs[6] = { -1, -1, -1, -1, -1, -1 };
				temp32 = 0;
				do {
					int fixedIndex = 0;
					do {
						fixedIndex = CudaNext(next, 7); // V�ӏ�
					} while(fixedIndex >= 6);

					if(ivs[fixedIndex] == -1)
					{
						ivs[fixedIndex] = 31;
						++temp32;
					}
				} while(temp32 < pokemon[1].flawlessIvs);

				// �̒l
				temp32 = 1;
				for(int i = 0; i < 6; ++i)
				{
					if(ivs[i] == 31)
					{
						if(pokemon[1].ivs[i] != 31)
						{
							temp32 = 0;
							break;
						}
					}
					else if(pokemon[1].ivs[i] != CudaNext(next, 0x1F))
					{
						temp32 = 0;
						break;
					}
				}
				if(temp32 == 0)
				{
					continue;
				}
			}

			// ����
			temp32 = 0;
			if(pokemon[0].abilityFlag == 3)
			{
				temp32 = CudaNext(seeds, 1);
			}
			else
			{
				do {
					temp32 = CudaNext(seeds, 3);
				} while(temp32 >= 3);
			}
			if((pokemon[0].ability >= 0 && pokemon[0].ability != temp32) || (pokemon[0].ability == -1 && temp32 >= 2))
			{
				continue;
			}
			temp32 = 0;
			if(pokemon[1].abilityFlag == 3)
			{
				temp32 = CudaNext(next, 1);
			}
			else
			{
				do {
					temp32 = CudaNext(next, 3);
				} while(temp32 >= 3);
			}
			if((pokemon[1].ability >= 0 && pokemon[1].ability != temp32) || (pokemon[1].ability == -1 && temp32 >= 2))
			{
				continue;
			}

			// ���ʒl
			if(!pokemon[0].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = CudaNext(seeds, 0xFF);
				} while(temp32 >= 253);
			}
			if(!pokemon[1].isNoGender)
			{
				temp32 = 0;
				do {
					temp32 = CudaNext(next, 0xFF);
				} while(temp32 >= 253);
			}

			// ���i
			temp32 = 0;
			do {
				temp32 = CudaNext(seeds, 0x1F);
			} while(temp32 >= 25);
			if(temp32 != pokemon[0].nature)
			{
				continue;
			}
			temp32 = 0;
			do {
				temp32 = CudaNext(next, 0x1F);
			} while(temp32 >= 25);
			if(temp32 != pokemon[1].nature)
			{
				continue;
			}
		}
		// ���ʂ���������
		int old = atomicAdd(pResultCount, 1);
		pResult[old] = temp64;
	}
	return;
}

// ������������
void Cuda5Initialize()
{
	// �z�X�g�������̊m��
	hipHostMalloc(&cu_HostInputCoefficientData, sizeof(_u32) * 0x8000);
	hipHostMalloc(&cu_HostInputSearchPattern, sizeof(_u32) * 0x4000);

	// �f�o�C�X�������̊m��
	hipMalloc(&pDeviceInput, sizeof(CudaInputMaster));
	hipMalloc(&pDeviceCoefficientData, sizeof(_u32) * 0x8000);
	hipMalloc(&pDeviceSearchPattern, sizeof(_u32) * 0x4000);
	hipMalloc(&pDeviceResultCount, sizeof(int));
	hipMalloc(&pDeviceResult, sizeof(_u64) * c_SizeResult);
}

// �f�[�^�Z�b�g
void Cuda5SetMasterData()
{
	// �z�X�g�f�[�^�̐ݒ�
	cu_HostInputMaster->constantTermVector[0] = (_u32)(g_ConstantTermVector >> 25);
	cu_HostInputMaster->constantTermVector[1] = (_u32)(g_ConstantTermVector & 0x1FFFFFFull);
	for(int i = 0; i < 64; ++i)
	{
		cu_HostInputMaster->answerFlag[i * 2] = (_u32)(g_AnswerFlag[i] >> 25);
		cu_HostInputMaster->answerFlag[i * 2 + 1] = (_u32)(g_AnswerFlag[i] & 0x1FFFFFFull);
	}
	for(int i = 0; i < 16 * 1024; ++i)
	{
		cu_HostInputCoefficientData[i * 2] = (_u32)(g_CoefficientData[i] >> 32);
		cu_HostInputCoefficientData[i * 2 + 1] = (_u32)(g_CoefficientData[i] & 0xFFFFFFFFull);
		cu_HostInputSearchPattern[i] = (_u32)g_SearchPattern[i];
	}

	// �f�[�^��]��
	hipMemcpy(pDeviceInput, cu_HostInputMaster, sizeof(CudaInputMaster), hipMemcpyHostToDevice);
	hipMemcpy(pDeviceCoefficientData, cu_HostInputCoefficientData, sizeof(_u32) * 0x8000, hipMemcpyHostToDevice);
	hipMemcpy(pDeviceSearchPattern, cu_HostInputSearchPattern, sizeof(_u32) * 0x4000, hipMemcpyHostToDevice);
}

// �v�Z
void Cuda5Process(_u32 param, int partition)
{
	// ���ʂ����Z�b�g
	*cu_HostResultCount = 0;
	hipMemcpy(pDeviceResultCount, cu_HostResultCount, sizeof(int), hipMemcpyHostToDevice);

	//�J�[�l��
	dim3 block(c_SizeBlockX, 1, 1);
	dim3 grid(c_SizeGridX / partition, 1, 1);
	kernel_calc << < grid, block >> > (pDeviceInput, pDeviceCoefficientData, pDeviceSearchPattern, pDeviceResultCount, pDeviceResult, param);

	//�f�o�C�X->�z�X�g�֌��ʂ�]��
	hipMemcpy(cu_HostResult, pDeviceResult, sizeof(_u64) * c_SizeResult, hipMemcpyDeviceToHost);
	hipMemcpy(cu_HostResultCount, pDeviceResultCount, sizeof(int), hipMemcpyDeviceToHost);
}

void Cuda5Finalize()
{
	//�f�o�C�X�������̊J��
	hipFree(pDeviceResult);
	hipFree(pDeviceResultCount);
	hipFree(pDeviceSearchPattern);
	hipFree(pDeviceCoefficientData);
	hipFree(pDeviceInput);
	//�z�X�g�������̊J��
	hipHostFree(cu_HostInputSearchPattern);
	hipHostFree(cu_HostInputCoefficientData);
}
